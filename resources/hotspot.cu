#include "hip/hip_runtime.h"
#ifdef RD_WG_SIZE_0_0
#define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
#define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
#define BLOCK_SIZE RD_WG_SIZE
#elif !defined(BLOCK_SIZE)
#define BLOCK_SIZE 16
#endif

/* maximum power density possible (say 300W for a 10mm x 10mm chip)	*/
#define MAX_PD	(3.0e6)
/* required precision in degrees	*/
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor	*/
#define FACTOR_CHIP	0.5

/* chip parameters	*/
float t_chip = 0.0005;
float chip_height = 0.016;
float chip_width = 0.016;
/* ambient temperature, assuming no package at all	*/
float amb_temp = 80.0;

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__device__ void calculate_temp(dim3 blockIdx,
                               int grid_cols,  //Col of grid
                               int grid_rows,  //Row of grid
                               int border_cols,  // border offset
                               int border_rows,  // border offset
                               float Cap,      //Capacitance
                               float Rx,
                               float Ry,
                               float Rz,
                               float step,
                               float time_elapsed,
                               const lightning::Matrix<float> power,   //power input
                               const lightning::Matrix<float> temp_src,    //temperature input/output
                               lightning::Matrix<float> temp_dst    //temperature input/output
                               ){

    __shared__ float temp_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float power_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float temp_t[BLOCK_SIZE][BLOCK_SIZE]; // saving temparary temperature result

    float amb_temp = 80.0;
    float step_div_Cap;
    float Rx_1,Ry_1,Rz_1;

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx=threadIdx.x;
    int ty=threadIdx.y;

    step_div_Cap=step/Cap;

    Rx_1=1/Rx;
    Ry_1=1/Ry;
    Rz_1=1/Rz;

    // each block finally computes result for a small block
    // after N iterations.
    // it is the non-overlapping small blocks that cover
    // all the input data

    // calculate the small block size
    int small_block_rows = BLOCK_SIZE-2;//EXPAND_RATE
    int small_block_cols = BLOCK_SIZE-2;//EXPAND_RATE

    // calculate the boundary for the block according to
    // the boundary of its small block
    int blkY = small_block_rows*by-border_rows;
    int blkX = small_block_cols*bx-border_cols;
    int blkYmax = blkY+BLOCK_SIZE-1;
    int blkXmax = blkX+BLOCK_SIZE-1;

    // calculate the global thread coordination
    int yidx = blkY+ty;
    int xidx = blkX+tx;

    // load data if it is within the valid input range
    int loadYidx=yidx, loadXidx=xidx;

    if(IN_RANGE(loadYidx, 0, grid_rows-1) && IN_RANGE(loadXidx, 0, grid_cols-1)){
        temp_on_cuda[ty][tx] = temp_src[loadYidx][loadXidx];  // Load the temperature data from global memory to shared memory
        power_on_cuda[ty][tx] = power[loadYidx][loadXidx];// Load the power data from global memory to shared memory
    }
    __syncthreads();

    // effective range within this block that falls within
    // the valid range of the input data
    // used to rule out computation outside the boundary.
    int validYmin = (blkY < 0) ? -blkY : 0;
    int validYmax = (blkYmax > grid_rows-1) ? BLOCK_SIZE-1-(blkYmax-grid_rows+1) : BLOCK_SIZE-1;
    int validXmin = (blkX < 0) ? -blkX : 0;
    int validXmax = (blkXmax > grid_cols-1) ? BLOCK_SIZE-1-(blkXmax-grid_cols+1) : BLOCK_SIZE-1;

    int N = ty-1;
    int S = ty+1;
    int W = tx-1;
    int E = tx+1;

    N = (N < validYmin) ? validYmin : N;
    S = (S > validYmax) ? validYmax : S;
    W = (W < validXmin) ? validXmin : W;
    E = (E > validXmax) ? validXmax : E;

    bool computed = false;
    int i = 0;
    if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
              IN_RANGE(ty, i+1, BLOCK_SIZE-i-2) &&  \
              IN_RANGE(tx, validXmin, validXmax) && \
              IN_RANGE(ty, validYmin, validYmax) ) {
        computed = true;
        temp_t[ty][tx] =   temp_on_cuda[ty][tx] + step_div_Cap * (power_on_cuda[ty][tx] +
                                                                  (temp_on_cuda[S][tx] + temp_on_cuda[N][tx] - 2.0*temp_on_cuda[ty][tx]) * Ry_1 +
                                                                  (temp_on_cuda[ty][E] + temp_on_cuda[ty][W] - 2.0*temp_on_cuda[ty][tx]) * Rx_1 +
                                                                  (amb_temp - temp_on_cuda[ty][tx]) * Rz_1);

    }
    __syncthreads();
    // update the global memory
    // after the last iteration, only threads coordinated within the
    // small block perform the calculation and switch on ``computed''
    if (computed){
        temp_dst[loadYidx][loadXidx] = temp_t[ty][tx];
    }
}